#include "hip/hip_runtime.h"
#include <cstdio>
#include "utils.h"
#include <vector>

constexpr size_t BLOCK_DIM = 32;

__global__ void matrix_multiply(float* A, float* B, float* C, size_t N) {

    // Calculate row and column of C to work on
    uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;

    float A_val = A[row * N + col];

    for (uint32_t i = 0; i < N; i++) {
        float B_val = B[row * N + i];
        float* C_dest = &C[row * N + i];
        atomicAdd(C_dest, A_val * B_val);
    }
}

int main(int argc, char* argv[]) {

    if (argc != 2) {
        printf("Usage: %s <N> \n", argv[0]);
        return 1;
    }

    size_t N = atoi(argv[1]);

    // This is just a demo, so enforce this for simplicity
    if (N % BLOCK_DIM != 0) {
        printf("N must be a multiple of %zu\n", BLOCK_DIM);
        return 1;
    }

    auto A_host = random_vector<float>(N * N);
    auto B_host = random_vector<float>(N * N);

    // Allocate memory on GPU
    float *A_device, *B_device, *C_device;
    hipMalloc(&A_device, N * N * sizeof(float));
    hipMalloc(&B_device, N * N * sizeof(float));
    hipMalloc(&C_device, N * N * sizeof(float));

    // Copy data to GPU
    hipMemcpy(A_device, A_host.data(), N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B_host.data(), N * N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid dimensions
    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
    dim3 grid_dim(N / BLOCK_DIM, N / BLOCK_DIM);

    printf("(%d x %d) grid of (%d x %d) blocks\n", grid_dim.x, grid_dim.y, block_dim.x, block_dim.y);

    // Time kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matrix_multiply<<<grid_dim, block_dim>>>(A_device, B_device, C_device, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    size_t FLOPs = 2 * N * N * N;
    printf("Time: %f ms, GFLOPs: %f\n", milliseconds, FLOPs / milliseconds / 1e6);

    // Copy result back to host
    auto C_host = std::vector<float>(N * N);
    hipMemcpy(C_host.data(), C_device, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);
}   